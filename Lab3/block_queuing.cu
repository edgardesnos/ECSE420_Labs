#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "io_helper.cuh"
#include "compare.cuh"

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define XNOR 5


/*
*
* Function to compute the output given a logical gate and the corresponding two input values.
*/
__device__ int gate_solver_parallel_block(int gateType, int inp1, int inp2) {
	int output;
	switch (gateType)
	{
	case AND:
		output = inp1 && inp2;
		break;
	case OR:
		output = inp1 || inp2;
		break;
	case NAND:
		output = !(inp1 && inp2);
		break;
	case NOR:
		output = !(inp1 || inp2);
		break;
	case XOR:
		output = (inp1 || inp2) && (!inp1 || !inp2);
		break;
	case XNOR:
		output = !((inp1 || inp2) && (!inp1 || !inp2));
		break;
	default:
		printf("Error: Gate not specified.\n");
		output = 0;
	}
	return output;
}

__global__ void blockQueuingKernel(int *nextLevelNodes, int *nodePtrs, int *nodeNeighbors, int *nodeInfo, int *currLevelNodes, int *numNextLevelNodes, float elementsPerThread)
{
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);

    // iterate over all the nodes assigned to the current thread
    for (int i = std::floor(idx * elementsPerThread); i < std::floor((idx + 1) * elementsPerThread); i++) {

        // obtain the node index from the current level nodes list
        int node = currLevelNodes[i];

        // iterate over all neighbors of the current level node selected
        for (int neighborIdx = nodePtrs[node]; neighborIdx < nodePtrs[node + 1]; neighborIdx++) {

            // obtain the neighbor node index from the node neighbors list
            int neighbor = nodeNeighbors[neighborIdx];

            // if this neighbor node has not yet been visited
            if (nodeInfo[neighbor*4] == 0) {

            	// set the node as visited
            	nodeInfo[neighbor*4] = 1;

            	// compute the node output
            	nodeInfo[neighbor*4 + 3] = gate_solver_parallel_block(nodeInfo[neighbor*4 + 1], nodeInfo[neighbor*4 + 2], nodeInfo[node*4 + 3]);

            	// store the node in nextLevelNodes -> make sure to use atomic addition instead of the ++ operator
                // Add to the memory block if space else to the global memory
                nextLevelNodes[atomicAdd(numNextLevelNodes, 1)] = neighbor;
            }
        }
    }
    // Once done with all elements, transfer block memory to global memory
}


// Helper function for using CUDA to perform global queuing
hipError_t blockQueuingHelper(
    int *nextLevelNodes, int *nodePtrs, int *nodeNeighbors, int *nodeInfo, int *currLevelNodes,
    int blockSize, int numBlock,
    int *numNextLevelNodes, int nodePtrs_size, int nodeNeighbors_size, int nodeInfo_size, int currLevelNodes_size
)
{
    int* dev_nextLevelNodes = 0;
    int* dev_nodePtrs = 0;
    int* dev_nodeNeighbors = 0;
    int* dev_nodeInfo = 0;
    int* dev_currLevelNodes = 0;
    int* dev_numNextLevelNodes;
    hipError_t cudaStatus;

    float elementsPerThread = (float) currLevelNodes_size / (blockSize * numBlock);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_nextLevelNodes, nodeInfo_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_nodePtrs, nodePtrs_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_nodeNeighbors, nodeNeighbors_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_nodeInfo, nodeInfo_size * 4 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_currLevelNodes, currLevelNodes_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_numNextLevelNodes, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_nodePtrs, nodePtrs, nodePtrs_size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_nodeNeighbors, nodeNeighbors, nodeNeighbors_size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_nodeInfo, nodeInfo, nodeInfo_size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_currLevelNodes, currLevelNodes, currLevelNodes_size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_numNextLevelNodes, numNextLevelNodes, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    blockQueuingKernel <<<numBlock, blockSize>>>(dev_nextLevelNodes, dev_nodePtrs, dev_nodeNeighbors, dev_nodeInfo, dev_currLevelNodes, dev_numNextLevelNodes, elementsPerThread);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "blockQueuingKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching blockQueuingKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(nextLevelNodes, dev_nextLevelNodes, nodeInfo_size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(nodeInfo, dev_nodeInfo, nodeInfo_size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(numNextLevelNodes, dev_numNextLevelNodes, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_nextLevelNodes);
    hipFree(dev_nodePtrs);
    hipFree(dev_nodeNeighbors);
    hipFree(dev_nodeInfo);
    hipFree(dev_currLevelNodes);
    
    return cudaStatus;
}


int main(int argc, char** argv)
{
    // validate input arguments
    if (argc != 9) {
        printf("Usage: ./block_queuing <blockSize> <numBlock> <inp1_file> <inp2_file> <inp3_file> <inp4_file> <nodeOutput_output_file> <nextLevelNodes_output_file>\n");
        exit(1);
    }

    // store the arguments
    int blockSize = std::stoi(argv[1]);
    int numBlock = std::stoi(argv[2]);
    char* inp1_filepath = argv[3];
    char* inp2_filepath = argv[4];
    char* inp3_filepath = argv[5];
    char* inp4_filepath = argv[6];
    char* nodeOutput_filepath = argv[7];
    char* nextLevelNodes_filepath = argv[8];

    // load the data from the files into arrays
    int nodePtrs_size, nodeNeighbors_size, nodeInfo_size, currLevelNodes_size;
    int* nodePtrs = input_reader(inp1_filepath, &nodePtrs_size);
    int* nodeNeighbors = input_reader(inp2_filepath, &nodeNeighbors_size);
    int* nodeInfo = input_reader_multiple(inp3_filepath, &nodeInfo_size);  // nodeInfo format -> visited, nodeGate, nodeInput, nodeOutput
    int* currLevelNodes = input_reader(inp4_filepath, &currLevelNodes_size);

    // initialize output variables
    int* nextLevelNodes = (int*)calloc(nodeInfo_size, sizeof(int));
    int numNextLevelNodes = 0;

    hipError_t cudaStatus = blockQueuingHelper(
        nextLevelNodes, nodePtrs, nodeNeighbors, nodeInfo, currLevelNodes,
        blockSize, numBlock,
        &numNextLevelNodes, nodePtrs_size, nodeNeighbors_size, nodeInfo_size, currLevelNodes_size
    );
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "blockQueuingHelper failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    // store the output in appropriate files
    int* nodeOutput = (int*)calloc(nodeInfo_size, sizeof(int));
    for (int i = 0; i < nodeInfo_size; i++) nodeOutput[i] = nodeInfo[i * 4 + 3];
    output_writer(nodeOutput_filepath, nodeOutput, nodeInfo_size);
    output_writer(nextLevelNodes_filepath, nextLevelNodes, numNextLevelNodes);

    // compare the results using the helper scripts provided
    printf("\nComparing the output files from the program with the solution files\n");
    printf("Comparing nodeOutput file: \n");
    compareFiles(nodeOutput_filepath, "./Lab3/Output/sol_nodeOutput.raw");
    printf("\nComparing nextLevelNodes file: \n");
    compareNextLevelNodeFiles(nextLevelNodes_filepath, "./Lab3/Output/sol_nextLevelNodes.raw");

    return 0;
}