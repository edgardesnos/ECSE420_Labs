
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <string>
#include <chrono>

int mainSequential(int argc, char** argv)
{

    if (argc != 2) {
        printf("Usage: ./sequential <num_iterations>");
        exit(1);
    }

    // load command args
    int T = std::stoi(argv[1]);

    const int N = 4;

    // Init u arrays
    float u2[N][N] = { 0 };
    float u1[N][N] = { 0 };
    float u[N][N] = { 0 };

    // constants
    float n = 0.0002;
    float p = 0.5;
    float G = 0.75;

    // Hit coordinates
    int hit_i = 2;
    int hit_j = 2;

    // Add the drum hit
    u1[hit_i][hit_j] = 1;

    printf("Size of grid: %d nodes\n", N*N);

    // initialize timer variables
    std::chrono::high_resolution_clock::time_point start, end;
    double exec_time, total_exec_time = 0.;

    // start the timer
    start = std::chrono::high_resolution_clock::now();

    for (int k = 0; k < T; k++) {
        // Interior elements
        for (int i = 1; i < N - 1; i++) {
            for (int j = 1; j < N - 1; j++) {
                u[i][j] = (p * (u1[i - 1][j] + u1[i + 1][j] + u1[i][j - 1] + u1[i][j + 1] - 4 * u1[i][j]) + 2 * u1[i][j] - (1 - n) * u2[i][j]) / (1 + n);
            }
        }
        // Side elements
        for (int i = 1; i < N - 1; i++) {
            u[0][i] = G * u[1][i];
            u[N - 1][i] = G * u[N - 2][i];
            u[i][0] = G * u[i][1];
            u[i][N - 1] = G * u[i][N - 2];
        }
        // Corner elements
        u[0][0] = G * u[1][0];
        u[N - 1][0] = G * u[N - 2][0];
        u[0][N - 1] = G * u[0][N - 2];
        u[N - 1][N - 1] = G * u[N - 1][N - 2];
        // Copy elements from u to u1 and u1 to u2
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                u2[i][j] = u1[i][j];
                u1[i][j] = u[i][j];
            }
        }
        printf("(0,0): %f (0,1): %f (0,2): %f (0,3): %f\n", u[0][0], u[0][1], u[0][2], u[0][3]);
        printf("(1,0): %f (1,1): %f (1,2): %f (1,3): %f\n", u[1][0], u[1][1], u[1][2], u[1][3]);
        printf("(2,0): %f (2,1): %f (2,2): %f (2,3): %f\n", u[2][0], u[2][1], u[2][2], u[2][3]);
        printf("(3,0): %f (3,1): %f (3,2): %f (3,3): %f\n", u[3][0], u[3][1], u[3][2], u[3][3]);
        printf("\n");
    }

    // end timer
    end = std::chrono::high_resolution_clock::now();

    // print the runtime
    exec_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count() / 1000000.;
    printf("\nThe runtime for sequential execution is: %f ms\n", exec_time);

    return 0;
}
