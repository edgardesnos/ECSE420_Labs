#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "data_helper.cuh"

#include <stdio.h>
#include <string>
#include <chrono>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define XNOR 5


float memsettime_explicit;
hipEvent_t start_explicit, stop_explicit;

hipError_t logicGateCudaExplicit(bool* output, bool* a, bool* b, char* gate, unsigned int size);

__global__ void logicGateKernelExplicit(bool* output, bool* a, bool* b, char* gate)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    switch (gate[i])
    {
    case AND:
        output[i] = a[i] && b[i];
        break;
    case OR:
        output[i] = a[i] || b[i];
        break;
    case NAND:
        output[i] = !(a[i] && b[i]);
        break;
    case NOR:
        output[i] = !(a[i] || b[i]);
        break;
    case XOR:
        output[i] = (a[i] || b[i]) && (!a[i] || !b[i]);
        break;
    case XNOR:
        output[i] = !((a[i] || b[i]) && (!a[i] || !b[i]));
        break;
    default:
        printf("Error: Gate not specified.\n");
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t logicGateCudaExplicit(bool* output, bool* a, bool* b, char* gate, unsigned int size)
{
    bool* dev_a = 0;
    bool* dev_b = 0;
    char* dev_gate = 0;
    bool* dev_output = 0;
    hipError_t cudaStatus;

    int num_blocks = (size / 1024) + 1;
    int num_threads = (size / num_blocks) + 1;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // initialize variables for timing data migration
    std::chrono::high_resolution_clock::time_point data_transfer_start, data_transfer_end;
    double data_migration_time;

    data_transfer_start = std::chrono::high_resolution_clock::now();
    // Allocate GPU buffers for four vectors (three input, one output)
    hipMalloc((void**)&dev_output, size * sizeof(bool));
    hipMalloc((void**)&dev_a, size * sizeof(bool));
    hipMalloc((void**)&dev_b, size * sizeof(bool));
    hipMalloc((void**)&dev_gate, size * sizeof(char));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(dev_gate, gate, size * sizeof(char), hipMemcpyHostToDevice);
    data_transfer_end = std::chrono::high_resolution_clock::now();
    
    // calculate and report the time taken for data migration
    data_migration_time = std::chrono::duration_cast<std::chrono::nanoseconds>(data_transfer_end - data_transfer_start).count() / 1000000.;
    printf("Time taken for data migration (host -> source) i.e., hipMalloc + hipMemcpy is: %f\n", data_migration_time);

    hipEventRecord(start_explicit, 0);
    logicGateKernelExplicit<<< num_blocks, num_threads >>>(dev_output, dev_a, dev_b, dev_gate);
    hipEventRecord(stop_explicit, 0);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "logicGateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching logicGateKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(output, dev_output, size * sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_output);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_gate);

    return cudaStatus;
}


int main(int argc, char* argv[])
{
    // perform input validation
    if (argc != 4) {
        printf("Usage: ./parallelal_explicit <input_file_name> <input_file_length> <output_file_name>\n");
        return 1;
    }

    // parse input arguments
    char* input_filename = argv[1];
    char* output_filename = argv[3];
    unsigned int size = std::stoi(argv[2]);

    // dynamically allocate memory for the arrays
    bool* a = (bool*)calloc(size, sizeof(bool));
    bool* b = (bool*)calloc(size, sizeof(bool));
    char* gate = (char*)calloc(size, sizeof(char));
    bool* output = (bool*)calloc(size, sizeof(bool));

    // load data from file into the arrays
    load_data(input_filename, a, b, gate, size);

    // initialize timer primitives
    hipEventCreate(&start_explicit);
    hipEventCreate(&stop_explicit);

    // Execute in parallel.
    hipError_t cudaStatus = logicGateCudaExplicit(output, a, b, gate, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "logicGateCuda failed!");
        return 1;
    }

    // save output data into a file
    save_data(output_filename, output, size);

    // report the CUDA kernel execution time
    hipEventSynchronize(stop_explicit);
    hipEventElapsedTime(&memsettime_explicit, start_explicit, stop_explicit);
    printf("\n *** CUDA kernel execution time with explicit memory allocation: %f *** \n", memsettime_explicit);
    hipEventDestroy(start_explicit);
    hipEventDestroy(stop_explicit);

    hipDeviceReset();
    return 0;
}