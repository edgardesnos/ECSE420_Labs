#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "data_helper.cuh"

#include <stdio.h>
#include <string>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define XNOR 5

float memsettime_unified;
hipEvent_t start_unified, stop_unified;

hipError_t logicGateCudaUnified(bool* output, bool* a, bool* b, char* gate, unsigned int size);

__global__ void logicGateKernelUnified(bool* output, bool* a, bool* b, char* gate)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    switch (gate[i])
    {
    case AND:
        output[i] = a[i] && b[i];
        break;
    case OR:
        output[i] = a[i] || b[i];
        break;
    case NAND:
        output[i] = !(a[i] && b[i]);
        break;
    case NOR:
        output[i] = !(a[i] || b[i]);
        break;
    case XOR:
        output[i] = (a[i] || b[i]) && (!a[i] || !b[i]);
        break;
    case XNOR:
        output[i] = !((a[i] || b[i]) && (!a[i] || !b[i]));
        break;
    default:
        printf("Error: Gate not specified.\n");
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t logicGateCudaUnified(bool* output, bool* a, bool* b, char* gate, unsigned int size)
{
    hipError_t cudaStatus;
    int num_blocks = (size / 1024) + 1;
    int num_threads = (size / num_blocks) + 1;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    hipEventRecord(start_unified, 0);
    logicGateKernelUnified<<<num_blocks, num_threads>>>(output, a, b, gate);
    hipEventRecord(stop_unified, 0);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "logicGateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching logicGateKernel!\n", cudaStatus);
        return cudaStatus;
    }
}


int main(int argc, char* argv[])
{
    // perform input validation
    if (argc != 4) {
        printf("Usage: ./parallelal_unified <input_file_name> <input_file_length> <output_file_name>\n");
        return 1;
    }

    // parse input arguments
    char* input_filename = argv[1];
    char* output_filename = argv[3];
    unsigned int size = std::stoi(argv[2]);

    // dynamically allocate memory for the arrays
    bool* a = (bool*)calloc(size, sizeof(bool));
    bool* b = (bool*)calloc(size, sizeof(bool));
    char* gate = (char*)calloc(size, sizeof(char));
    bool* output = (bool*)calloc(size, sizeof(bool));

    // initialize timer primitives
    hipEventCreate(&start_unified);
    hipEventCreate(&stop_unified);

    // allocate unified memory
    hipMallocManaged((void**)&output, size * sizeof(bool));
    hipMallocManaged((void**)&a, size * sizeof(bool));
    hipMallocManaged((void**)&b, size * sizeof(bool));
    hipMallocManaged((void**)&gate, size * sizeof(char));

    // load data from file into the arrays
    load_data(input_filename, a, b, gate, size);

    // Execute in parallel.
    hipError_t cudaStatus = logicGateCudaUnified(output, a, b, gate, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "logicGateCuda failed!");
        return 1;
    }

    // save output data into a file
    save_data(output_filename, output, size);

    // report the CUDA kernel execution time
    hipEventSynchronize(stop_unified);
    hipEventElapsedTime(&memsettime_unified, start_unified, stop_unified);
    printf("\n *** CUDA kernel execution time with unified memory allocation: %f *** \n", memsettime_unified);
    hipEventDestroy(start_unified);
    hipEventDestroy(stop_unified);

    hipFree(output);
    hipFree(a);
    hipFree(b);
    hipFree(gate);

    hipDeviceReset();
    return 0;
}