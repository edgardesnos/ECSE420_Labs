﻿
#include "hip/hip_runtime.h"
#include ""
#include "data_helper.cuh"

#include <stdio.h>
#include <string>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define XNOR 5

bool gateOutput(bool a, bool b, char gate);


void logicGateSequential(bool* output, bool* a, bool* b, char* gate, int length) 
{
    for (int i=0; i<length; i++) 
    {
        output[i] = gateOutput(a[i], b[i], gate[i]);
    }
}

bool gateOutput(bool a, bool b, char gate)
{
    bool output = 0;
    switch (gate)
    {
    case AND:
        output = a && b;
        break;
    case OR:
        output = a || b;
        break;
    case NAND:
        output = !(a && b);
        break;
    case NOR:
        output = !(a || b);
        break;
    case XOR:
        output = (a || b) && (!a || !b);
        break;
    case XNOR:
        output = !((a || b) && (!a || !b));
        break;
    default:
        printf("Error: Gate not specified.\n");
    }
    return output;
}


int main(int argc, char* argv[])
{
    if (argc != 4) {
        printf("Usage: ./sequential <input_file_name> <input_file_length> <output_file_name>\n");
        return 1;
    }

    char* input_filename = argv[1];
    char* output_filename = argv[3];
    unsigned int size = std::stoi(argv[2]);

    // dynamically allocate memory for the arrays
    bool* a = (bool*)calloc(size, sizeof(bool));
    bool* b = (bool*)calloc(size, sizeof(bool));
    char* gate = (char*)calloc(size, sizeof(char));
    bool* output = (bool*)calloc(size, sizeof(bool));

    // load data from file into the arrays
    load_data(input_filename, a, b, gate, size);

    //Execute sequentially
    logicGateSequential(output, a, b, gate, size);

    printf("Output array : {%d,%d,%d,%d}\n",output[0], output[1], output[2], output[3]);

    // save output data into a file
    save_data(output_filename, output, size);

    return 0;
}
