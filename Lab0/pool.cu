#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "img_helper.cuh"
#include "gputimer.h"

#include <stdio.h>
#include <math.h>
#include <string>

#define MAX_MSE 0.00001f


//hipError_t pool(int* out, const int* in, unsigned int size);
hipError_t pool(unsigned char* image_out, unsigned char* image_in, unsigned int width, unsigned int size, unsigned int threads_per_block, struct GpuTimer* timer, float* timeElapsed);

__global__ void poolKernel(unsigned char* out, unsigned char* in, unsigned int width, unsigned int size)
{
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    int x = (idx/4) % width;
    int y = (int)idx / (width * 4);
    int out_idx = (idx / 4) + x + (idx % 4);
    if (y % 2 == 0 && idx%8 < 4 && idx < size && out_idx < size/4) {
        out[out_idx] = max(max(in[idx], in[idx + 4]), max(in[idx + (4*width)], in[idx + (4*width) + 4]));
    }
}


// Helper function for using CUDA to perform 2 x 2 max pooling on a single image channel.
hipError_t pool(unsigned char* out, unsigned char* in, unsigned int width, unsigned int size, unsigned int threads_per_block, struct GpuTimer* timer, float* timeElapsed)
{
    unsigned char* dev_in = 0;
    unsigned char* dev_out = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_out, (size / 4) * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_in, size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
  
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_in, in, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU and time it
    timer->Start();
    poolKernel << < (size + threads_per_block - 1) / threads_per_block, threads_per_block >> > (dev_out, dev_in, width, size);
    timer->Stop();

    // record the computation time
    *timeElapsed = timer->Elapsed();

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, (size / 4) * sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_out);
    hipFree(dev_in);

    return cudaStatus;
}


float get_MSE(char* input_filename_1, char* input_filename_2)
{
    unsigned error1, error2;
    unsigned char* image1, * image2;
    unsigned width1, height1, width2, height2;

    error1 = lodepng_decode32_file(&image1, &width1, &height1, input_filename_1);
    error2 = lodepng_decode32_file(&image2, &width2, &height2, input_filename_2);
    if (error1) printf("error %u: %s\n", error1, lodepng_error_text(error1));
    if (error2) printf("error %u: %s\n", error2, lodepng_error_text(error2));
    if (width1 != width2) printf("images do not have same width\n");
    if (height1 != height2) printf("images do not have same height\n");

    // process image
    float im1, im2, diff, sum, MSE;
    sum = 0;
    for (int i = 0; i < width1 * height1; i++) {
        im1 = (float)image1[i];
        im2 = (float)image2[i];
        diff = im1 - im2;
        sum += diff * diff;
    }
    MSE = sqrt(sum) / (width1 * height1);

    free(image1);
    free(image2);

    return MSE;
}


int main(int argc, char** argv) {
    if (argc != 5) {
        printf("Usage: ./pool <input_image> <output_image> <gold_standard_output_filename> <num_threads>");
        exit(1);
    }

    // load command args
    char* input_filename = argv[1];
    char* output_filename = argv[2];
    char* gold_standard_output_filename = argv[3];
    unsigned int NUM_THREADS = std::stoi(argv[4]);

    if (NUM_THREADS > 1024) {
        printf("Maximum number of threads can be 1024. Aborting...");
        exit(1);
    }

    float totalTimeElapsed = 0.0;

    // repeat the operation 10 times
    for (int i = 0; i < 10; i++) {
        // initialize timer
        float timeElapsed = 0.0;
        struct GpuTimer* timer = new GpuTimer();

        // load the image
        unsigned int width, height, size;
        unsigned char* image = { 0 };
        //unsigned char* image = (unsigned char*) malloc(size * sizeof(unsigned char));
        load_image(input_filename, &image, &width, &height, &size);

        // perform pooling
        unsigned char* image_out = (unsigned char*)malloc((size / 4) * sizeof(unsigned char));
        pool(image_out, image, width, size, NUM_THREADS, timer, &timeElapsed);
        printf("Pooling computation time: %f\n", timeElapsed);
        totalTimeElapsed += timeElapsed;

        // save the image
        save_image(output_filename, image_out, width / 2, height / 2);

        // compare the generated image to the provided correct output image
        printf("MSE between the generated output and given output: %f\n", get_MSE(output_filename, gold_standard_output_filename));

        // free memory
        free(image);
        free(image_out);
    }

    // obtain the average total time for performing the pooling operation
    printf("Average computation time for preforming pooling: %f\n", totalTimeElapsed/10.0);
}